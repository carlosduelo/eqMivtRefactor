#include "hip/hip_runtime.h"
/*
Author: Carlos Duelo Serrano 
Company: Cesvima

Notes:

*/

#include "octree_cuda.h"

#ifndef DEVICE_CODE
#include <../src/textures.cu>
#endif

#include "cuda_help.h"
#include "mortonCodeUtil.h"

#include "cutil_math.h"

#include <iostream>
#include <fstream>

namespace eqMivt
{
#ifndef DEVICE_CODE
inline __device__ float3 _cuda_BoxToCoordinates(int3 pos, int3 realDim)
{
	float3 r;
	r.x = pos.x >= realDim.x ? tex1Dfetch(xgrid, CUBE_INC + realDim.x-1) + pos-x - realDim.x : tex1Dfetch(xgrid, CUBE_INC + pos.x);
	r.y = pos.y >= realDim.y ? tex1Dfetch(ygrid, CUBE_INC + realDim.y-1) + pos-x - realDim.x : tex1Dfetch(ygrid, CUBE_INC + pos.y);
	r.z = pos.z >= realDim.z ? tex1Dfetch(zgrid, CUBE_INC + realDim.z-1) + pos-x - realDim.x : tex1Dfetch(zgrid, CUBE_INC + pos.z);

	return r;
}
#endif
/*
 **********************************************************************************************
 ****** GPU Octree functions ******************************************************************
 **********************************************************************************************
 */

__device__ inline bool _cuda_checkRangeGrid(index_node_t * elements, index_node_t index, int min, int max)
{
		return elements[max] >= index && elements[min] <= index;
}

__device__ int _cuda_binary_search_closer_Grid(index_node_t * elements, index_node_t index, int min, int max)
{
	int middle = 0;
	while(1)
	{
		int diff 	= max-min;
		middle	= min + (diff / 2);
		if (middle % 2 == 1) middle--;

		if (diff <= 1) return middle;
		if (elements[middle+1] >= index && elements[middle] <= index) return middle;
		if (index < elements[middle])
			max = middle-1;
		else //(index > elements[middle+1])
			min = middle + 2;
	}
}

__device__  bool _cuda_searchSecuentialGrid(index_node_t * elements, index_node_t index, int min, int max)
{
	for(int i=min; i<max; i+=2)
	{
		if (elements[i] > index)
			return false;
		if (elements[i+1] >= index && elements[i] <= index)
			return true;
	}

	return false;
}

__device__ bool _cuda_RayAABB(index_node_t index, float3 origin, float3 dir,  float * tnear, float * tfar, int nLevels, int3 realDim)
{
	int3 minBoxC;
	int3 maxBoxC;
	int level;
	minBoxC = getMinBoxIndex(index, &level, nLevels); 
	if (minBoxC.x >= realDim.x || minBoxC.y >= realDim.y || minBoxC.y >= realDim.y)
		return false;
	int dim = (1<<(nLevels-level));
	maxBoxC.x = dim + minBoxC.x;
	maxBoxC.y = dim + minBoxC.y;
	maxBoxC.z = dim + minBoxC.z;
	float3 minBox = _cuda_BoxToCoordinates(minBoxC, realDim);
	float3 maxBox = _cuda_BoxToCoordinates(maxBoxC, realDim);

	float tmin, tmax, tymin, tymax, tzmin, tzmax;
	float divx = 1.0f / dir.x;
	if (divx >= 0.0f)
	{
		tmin = (minBox.x - origin.x)*divx;
		tmax = (maxBox.x - origin.x)*divx;
	}
	else
	{
		tmin = (maxBox.x - origin.x)*divx;
		tmax = (minBox.x - origin.x)*divx;
	}
	float divy = 1.0f / dir.y;
	if (divy >= 0.0f)
	{
		tymin = (minBox.y - origin.y)*divy;
		tymax = (maxBox.y - origin.y)*divy;
	}
	else
	{
		tymin = (maxBox.y - origin.y)*divy;
		tymax = (minBox.y - origin.y)*divy;
	}

	if ( (tmin > tymax) || (tymin > tmax) )
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	float divz = 1.0f / dir.z;
	if (divz >= 0.0f)
	{
		tzmin = (minBox.z - origin.z)*divz;
		tzmax = (maxBox.z - origin.z)*divz;
	}
	else
	{
		tzmin = (maxBox.z - origin.z)*divz;
		tzmax = (minBox.z - origin.z)*divz;
	}

	if ( (tmin > tzmax) || (tzmin > tmax) )
		return false;

	if (tzmin > tmin)
		tmin = tzmin;
	if (tzmax < tmax)
		tmax = tzmax;

	if (tmin<0.0f)
	 	*tnear=0.0f;
	else
		*tnear=tmin;
	*tfar=tmax;

	return *tnear < *tfar;
}

__device__ bool _cuda_RayAABB2(float3 origin, float3 dir,  float * tnear, float * tfar, int nLevels, int3 minBoxC, int level, int3 realDim)
{
	if (minBoxC.x >= realDim.x || minBoxC.y >= realDim.y || minBoxC.y >= realDim.y)
		return false;

	int3 maxBoxC;
	int dim = (1<<(nLevels-level));
	maxBoxC.x = dim + minBoxC.x;
	maxBoxC.y = dim + minBoxC.y;
	maxBoxC.z = dim + minBoxC.z;
	float3 minBox = _cuda_BoxToCoordinates(minBoxC, realDim);
	float3 maxBox = _cuda_BoxToCoordinates(maxBoxC, realDim);

	float tmin, tmax, tymin, tymax, tzmin, tzmax;
	float divx = 1.0f / dir.x;
	if (divx >= 0.0f)
	{
		tmin = (minBox.x - origin.x)*divx;
		tmax = (maxBox.x - origin.x)*divx;
	}
	else
	{
		tmin = (maxBox.x - origin.x)*divx;
		tmax = (minBox.x - origin.x)*divx;
	}
	float divy = 1.0f / dir.y;
	if (divy >= 0.0f)
	{
		tymin = (minBox.y - origin.y)*divy;
		tymax = (maxBox.y - origin.y)*divy;
	}
	else
	{
		tymin = (maxBox.y - origin.y)*divy;
		tymax = (minBox.y - origin.y)*divy;
	}

	if ( (tmin > tymax) || (tymin > tmax) )
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	float divz = 1.0f / dir.z;
	if (divz >= 0.0f)
	{
		tzmin = (minBox.z - origin.z)*divz;
		tzmax = (maxBox.z - origin.z)*divz;
	}
	else
	{
		tzmin = (maxBox.z - origin.z)*divz;
		tzmax = (minBox.z - origin.z)*divz;
	}

	if ( (tmin > tzmax) || (tzmin > tmax) )
		return false;
	if (tzmin > tmin)
		tmin = tzmin;
	if (tzmax < tmax)
		tmax = tzmax;

	if (fabsf(tmax -tmin) < EPS)
		return false;

	if (tmin<0.0f)
	 	*tnear=0.0f;
	else
		*tnear=tmin;

	*tfar=tmax;

	return *tnear < *tfar;

}

__device__ bool _cuda_searchNextChildrenValidAndHit(index_node_t * elements, int size, int3 realDim, float3 origin, float3 ray, index_node_t father, float cTnear, float cTfar, int nLevels, int level, int3 minB, index_node_t * child, float * childTnear, float * childTfar)
{
#if 0
	int dimC = 1 << (nLevels - level);
	int dimF = dimC << 1;
	float3 minBoxC = _cuda_BoxToCoordinates(minB, realDim);	
	float3 midBoxC = _cuda_BoxToCoordinates(minB + make_int3(dimC, dimC, dimC), realDim);	
	float3 maxBoxC = _cuda_BoxToCoordinates(minB + make_int3(dimF, dimF, dimF), realDim);	
	float3 tmin, tmid, tmax;
	float3 div = make_float3(1.0f / ray.x, 1.0f / ray.y, 1.0f / ray.z); 
	index_node_t mask = 0;

	if (div.x >= 0)
	{
		tmin.x = minBoxC.x - origin.x;
		tmid.x = midBoxC.x - origin.x;
		tmax.x = maxBoxC.x - origin.x;
	}
	else
	{
		tmin.x = maxBoxC.x - origin.x;
		tmid.x = midBoxC.x - origin.x;
		tmax.x = minBoxC.x - origin.x;
		mask |= 4;
	}
	if (div.y >= 0)
	{
		tmin.y = minBoxC.y - origin.y;
		tmid.y = midBoxC.y - origin.y;
		tmax.y = maxBoxC.y - origin.y;
	}                     
	else                  
	{                     
		tmin.y = maxBoxC.y - origin.y;
		tmid.y = midBoxC.y - origin.y;
		tmax.y = minBoxC.y - origin.y;
		mask |= 2;
	}
	if (div.z >= 0)
	{
		tmin.z = minBoxC.z - origin.x;
		tmid.z = midBoxC.z - origin.x;
		tmax.z = maxBoxC.z - origin.x;
	}                     
	else                  
	{                     
		tmin.z = maxBoxC.z - origin.z;
		tmid.z = midBoxC.z - origin.z;
		tmax.z = minBoxC.z - origin.z;
		mask |= 1;
	}

	tmin = tmin * div;
	tmid = tmid * div;
	tmax = tmax * div;

	index_node_t c = 0;
	index_node_t childrenID = father << 3;
	unsigned int closer1 = 0;
	unsigned int closer8 = size;

	if (size != 2)
	{
		closer1 =  _cuda_binary_search_closer_Grid(elements , childrenID , 0, size-1);
	}

	float text = 0.0f;
	float tent = 0.0f;
	float closer = 0x7ff0000000000000;	//infinity
	bool find = false;

	c |= tent > tmid.x && tent < tmax.x ? 4 : 0;
	c |= tent > tmid.y && tent < tmax.y ? 2 : 0;
	c |= tent > tmid.z && tent < tmax.z ? 1 : 0;

	switch(c)
	{
		case 0:
				tent = fmaxf(tmin.x, fmaxf(tmin.y, tmin.z));
				text = fminf(tmid.x, fminf(tmid.y, tmid.z));
			break;  
		case 1:
				tent = fmaxf(tmin.x, fmaxf(tmin.y, tmid.z));
				text = fminf(tmid.x, fminf(tmid.y, tmax.z));
			break;  
		case 2:
				tent = fmaxf(tmin.x, fmaxf(tmid.y, tmin.z));
				text = fminf(tmid.x, fminf(tmax.y, tmid.z));
			break;  
		case 3:
				tent = fmaxf(tmin.x, fmaxf(tmid.y, tmid.z));
				text = fminf(tmid.x, fminf(tmax.y, tmax.z));
			break;  
		case 4:
				tent = fmaxf(tmid.x, fmaxf(tmin.y, tmin.z));
				text = fminf(tmax.x, fminf(tmid.y, tmid.z));
			break;  
		case 5:
				tent = fmaxf(tmid.x, fmaxf(tmin.y, tmid.z));
				text = fminf(tmax.x, fminf(tmid.y, tmax.z));
			break;  
		case 6:
				tent = fmaxf(tmid.x, fmaxf(tmid.y, tmin.z));
				text = fminf(tmax.x, fminf(tmax.y, tmid.z));
			break;  
		case 7:
				tent = fmaxf(tmid.x, fmaxf(tmid.y, tmid.z));
				text = fminf(tmax.x, fminf(tmax.y, tmax.z));
			break;  
	}

	do
	{

		if (tent < text && 
			tent >= cTnear && 
			tent <= closer && 
			_cuda_searchSecuentialGrid(elements, childrenID | (c^mask), closer1, closer8))
		{
			*child = childrenID | (c ^ mask);
			*childTnear = tent;
			*childTfar = text;
			closer = tent;
			find = true;
		}

		float te = 0.0f;
		switch(c)
		{
			case 0:
				te = fminf(tmid.x, fminf(tmid.y, tmid.z));
				if (te == tmid.x)
				{
					c=4;
					tent = fmaxf(tmid.x, fmaxf(tmin.y, tmin.z));
					text = fminf(tmax.x, fminf(tmid.y, tmid.z));
				}
				else if (te == tmid.y)
				{
					c=2;
					tent = fmaxf(tmin.x, fmaxf(tmid.y, tmin.z));
					text = fminf(tmid.x, fminf(tmax.y, tmid.z));
				}
				else if (te == tmid.z)
				{
					c=1;
					tent = fmaxf(tmin.x, fmaxf(tmin.y, tmid.z));
					text = fminf(tmid.x, fminf(tmid.y, tmax.z));
				}
				break;  
			case 1:	
				te = fminf(tmid.x, fminf(tmid.y, tmax.z));
				if (te == tmid.x)
				{
					c=5;
					tent = fmaxf(tmid.x, fmaxf(tmin.y, tmid.z));
					text = fminf(tmax.x, fminf(tmid.y, tmax.z));
				}
				else if (te == tmid.y)
				{
					c=3;
					tent = fmaxf(tmin.x, fmaxf(tmid.y, tmid.z));
					text = fminf(tmid.x, fminf(tmax.y, tmax.z));
				}
				else
					c=8;
				break;  
			case 2:	
				te = fminf(tmid.x, fminf(tmax.y, tmid.z));
				if (te == tmid.x) 
				{
					c=6;
					tent = fmaxf(tmid.x, fmaxf(tmid.y, tmin.z));
					text = fminf(tmax.x, fminf(tmax.y, tmid.z));
				}
				else if (te == tmid.z)
				{
					c=3;
					tent = fmaxf(tmin.x, fmaxf(tmid.y, tmid.z));
					text = fminf(tmid.x, fminf(tmax.y, tmax.z));
				}
				else
					c=8;
				break;  
			case 3:	
				te = fminf(tmid.x, fminf(tmax.y, tmax.z));
				if (te == tmid.x) 
				{
					c=7;
					tent = fmaxf(tmid.x, fmaxf(tmid.y, tmid.z));
					text = fminf(tmax.x, fminf(tmax.y, tmax.z));
				}
				else 
					c=8;
				break;  
			case 4:	
				te = fminf(tmax.x, fminf(tmid.y, tmid.z));
				if (te == tmid.y)
				{
					c=6;
					tent = fmaxf(tmid.x, fmaxf(tmid.y, tmin.z));
					text = fminf(tmax.x, fminf(tmax.y, tmid.z));
				}
				else if (te == tmid.z)
				{
					c=5;
					tent = fmaxf(tmid.x, fmaxf(tmin.y, tmid.z));
					text = fminf(tmax.x, fminf(tmid.y, tmax.z));
				}
				else
					c=8;
				break;  
			case 5:	
				te = fminf(tmax.x, fminf(tmid.y, tmax.z));
				if (te == tmid.y)
				{
					c=7;
					tent = fmaxf(tmid.x, fmaxf(tmid.y, tmid.z));
					text = fminf(tmax.x, fminf(tmax.y, tmax.z));
				}
				else 
					c=8;
				break;  
			case 6:	
				te = fminf(tmax.x, fminf(tmax.y, tmid.z));
				if (te == tmid.z) 
				{
					c=7;
					tent = fmaxf(tmid.x, fmaxf(tmid.y, tmid.z));
					text = fminf(tmax.x, fminf(tmax.y, tmax.z));
				}
				else
					c=8;
				break;  
			case 7:
				c = 8;
				break;  
		}
	}
	while(c < 8);

	return find;

#else
	index_node_t childrenID = father << 3;
	int dim = (1<<(nLevels-level));
	float closer = 0x7ff0000000000000;	//infinity
	bool find = false;
	float childTnearT = 0xfff0000000000000; // -infinity
	float childTfarT  = 0xfff0000000000000; // -infinity
	int3 minBox = minB;

	if (size==2)
	{
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.x+=dim;
		minBox.y-=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
	}
	else
	{
		unsigned int closer1 = _cuda_binary_search_closer_Grid(elements, childrenID,   0, size-1);
		unsigned int closer8 = size;

		if (closer8 >= size)
			closer8 = size-1;

		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.x+=dim;
		minBox.y-=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
	}

	return find;
	#endif
}

__device__ int3 _cuda_updateCoordinatesGrid(int maxLevel, int cLevel, index_node_t cIndex, int nLevel, index_node_t nIndex, int3 minBox)
{
	if ( 0 == nIndex)
	{
		return make_int3(0,0,0);
	}
	else if (cLevel < nLevel)
	{
		int dim = 1 << (maxLevel-nLevel);
		minBox.z +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.y +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.x +=  (nIndex & 0x1) * dim;
		return minBox;

	}
	else if (cLevel > nLevel)
	{
		return	getMinBoxIndex2(nIndex, nLevel, maxLevel);
	}
	else
	{
		int dim = 1 << (maxLevel-nLevel);
		minBox.z +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.y +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.x +=  (nIndex & 0x1) * dim;
		minBox.z -=  (cIndex & 0x1) * dim; cIndex>>=1;
		minBox.y -=  (cIndex & 0x1) * dim; cIndex>>=1;
		minBox.x -=  (cIndex & 0x1) * dim;
		return minBox;
	}
}

__device__ bool _cuda_octreeIteration(index_node_t ** octree, int * sizes, float3 origin, float3 ray, int nLevels, int finalLevel, visibleCube_t * indexNode, int3 realDim, float * currentTnear, float * currentTfar)
{
	*currentTnear	= 0.0f;
	*currentTfar	= 0.0f;
	index_node_t 	current			= indexNode->id == 0 ? 1 : indexNode->id;
	int				currentLevel	= 0;

	// Update tnear and tfar
	if (!_cuda_RayAABB(current, origin, ray,  currentTnear, currentTfar, nLevels, realDim) || (*currentTfar) < 0.0f)
	{
		// NO CUBE FOUND
		indexNode->state = CUDA_NOCUBE;
		return false;
	}
	if (current != 1)
	{
		current  >>= 3;
		currentLevel = finalLevel - 1;
		*currentTnear = *currentTfar;
	}

	int3		minBox 		= getMinBoxIndex2(current, currentLevel, nLevels);

	while(1)
	{
		if (currentLevel == finalLevel)
		{
			indexNode->id = current;
			indexNode->state = CUDA_CUBE;
			return true;
		}

		// Get fitst child >= currentTnear away
		index_node_t	child;
		float			childTnear;
		float			childTfar;
		if (_cuda_searchNextChildrenValidAndHit(octree[currentLevel+1], sizes[currentLevel+1], realDim, origin, ray, current, *currentTnear, *currentTfar, nLevels, currentLevel+1, minBox, &child, &childTnear, &childTfar))
		{
			minBox = _cuda_updateCoordinatesGrid(nLevels, currentLevel, current, currentLevel + 1, child, minBox);
			current = child;
			currentLevel++;
			*currentTnear = childTnear;
			*currentTfar = childTfar;
		}
		else if (current == 1) 
		{
			indexNode->state = CUDA_NOCUBE;
			return false;
		}
		else
		{
			minBox = _cuda_updateCoordinatesGrid(nLevels, currentLevel, current, currentLevel - 1, current >> 3, minBox);
			current >>= 3;
			currentLevel--;
			*currentTnear = *currentTfar;
		}

	}
}

__device__ bool _cuda_rayCaster(float3 origin, float3  LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int numRays, float iso, visibleCube_t * cube, int levelO, int levelC, int nLevel, float maxHeight, int3 realDim, float * r, float * g, float * b, float * screen, int offset, float * data)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < numRays)
	{
		tid += offset;
		float tnear;
		float tfar;
		// To do test intersection real cube position
		int3 minBox = getMinBoxIndex2(cube->id, levelO, nLevel);
		int dim = 1 << (nLevel-levelO);
		//int dim = powf(2,nLevel-levelO);
		int3 maxBox = minBox + make_int3(dim,dim,dim);
		float3 minBoxC = _cuda_BoxToCoordinates(minBox, realDim);
		float3 maxBoxC = _cuda_BoxToCoordinates(maxBox, realDim);

		int i = tid % pvpW;
		int j = tid / pvpW;

		float3 ray = LB - origin;
		ray += (j*h)*up + (i*w)*right;
		ray = normalize(ray);

		if  (_cuda_RayAABB(origin, ray,  &tnear, &tfar, minBoxC, maxBoxC))
		{
			// To ray caster is needed bigger cube, so add cube inc
			int3 minBoxD = getMinBoxIndex2(cube->id >> (3*(levelO - levelC)), levelC, nLevel) - make_int3(CUBE_INC, CUBE_INC, CUBE_INC);
			int3 dimD;
			dimD.x = (1 << (nLevel-levelC)) + 2*CUBE_INC;
			dimD.y = (1 << (nLevel-levelC)) + 2*CUBE_INC;
			dimD.z = (1 << (nLevel-levelC)) + 2*CUBE_INC;

			float3 Xnear = origin + tnear * ray;

			int3 pos = make_int3(	_cuda_searchCoordinateX(Xnear.x, minBox.x - 1, maxBox.x+1),
									_cuda_searchCoordinateY(Xnear.y, minBox.y - 1, maxBox.y+1),
									_cuda_searchCoordinateZ(Xnear.z, minBox.z - 1, maxBox.z+1));

			bool hit = false;
			float3 Xfar = Xnear;
			float3 Xnew = Xnear;
			bool primera 	= true;
			float ant		= 0.0f;
			float sig		= 0.0f;

			while (!hit &&
				(minBox.x-1 <= pos.x && pos.x <= maxBox.x) &&
				(minBox.y-1 <= pos.y && pos.y <= maxBox.y) &&
				(minBox.z-1 <= pos.z && pos.z <= maxBox.z))
			{
				if (pos.x >= 0 && pos.y >= 0 && pos.z >= 0 && pos.x < realDim.x-1 && pos.y < realDim.y-1 && pos.z < realDim.z-1)
				{
					float3 xyz = make_float3(	pos.x + ((Xnear.x - tex1Dfetch(xgrid, pos.x + CUBE_INC)) / (tex1Dfetch(xgrid, pos.x+1 + CUBE_INC) - tex1Dfetch(xgrid, pos.x + CUBE_INC))),
												pos.y + ((Xnear.y - tex1Dfetch(ygrid, pos.y + CUBE_INC)) / (tex1Dfetch(ygrid, pos.y+1 + CUBE_INC) - tex1Dfetch(ygrid, pos.y + CUBE_INC))),
												pos.z + ((Xnear.z - tex1Dfetch(zgrid, pos.z + CUBE_INC)) / (tex1Dfetch(zgrid, pos.z+1 + CUBE_INC) - tex1Dfetch(zgrid, pos.z + CUBE_INC))));
					
					if (primera)
					{
						ant = getElementInterpolateGrid(xyz, data, minBoxD, dimD);
						Xfar = Xnear;
						primera = false;
					}
					else
					{
						sig = getElementInterpolateGrid(xyz, data, minBoxD, dimD);

						if (( ((iso-ant)<0.0f) && ((iso-sig)<0.0f)) || ( ((iso-ant)>0.0f) && ((iso-sig)>0.0)))
						{
							ant = sig;
							Xfar=Xnear;
						}
						else
						{
							float a = (iso-ant)/(sig-ant);
							Xnew = Xfar*(1.0f-a)+ Xnear*a;
							hit = true;
						}
					}
				}

				// Update Xnear
				Xnear += ((fminf(fabs(tex1Dfetch(xgrid, pos.x+1 + CUBE_INC) - tex1Dfetch(xgrid, pos.x + CUBE_INC)), fminf(fabs(tex1Dfetch(ygrid, pos.y+1 + CUBE_INC) - tex1Dfetch(ygrid, pos.y + CUBE_INC)),fabs( tex1Dfetch(zgrid, pos.z+1 + CUBE_INC) - tex1Dfetch(zgrid, pos.z + CUBE_INC))))) / 3.0f) * ray;

				// Get new pos
				while((minBox.x-2 <= pos.x && pos.x <= maxBox.x + 1) &&  !(tex1Dfetch(xgrid, pos.x + CUBE_INC) <= Xnear.x && Xnear.x < tex1Dfetch(xgrid, pos.x+1 + CUBE_INC)))
					pos.x = ray.x < 0 ? pos.x - 1 : pos.x +1;
				while((minBox.y-2 <= pos.y && pos.y <= maxBox.y + 1) &&!(tex1Dfetch(ygrid, pos.y + CUBE_INC) <= Xnear.y && Xnear.y < tex1Dfetch(ygrid, pos.y+1 + CUBE_INC)))
					pos.y = ray.y < 0 ? pos.y - 1 : pos.y +1;
				while((minBox.z-2 <= pos.z && pos.z <= maxBox.z + 1) &&!(tex1Dfetch(zgrid, pos.z + CUBE_INC) <= Xnear.z && Xnear.z < tex1Dfetch(zgrid, pos.z+1 + CUBE_INC)))
					pos.z = ray.z < 0 ? pos.z - 1 : pos.z +1;
			}

			if (hit)
			{
				pos = make_int3(	_cuda_searchCoordinateX(Xnew.x, minBox.x - 1, maxBox.x+1),
									_cuda_searchCoordinateY(Xnew.y, minBox.y - 1, maxBox.y+1),
									_cuda_searchCoordinateZ(Xnew.z, minBox.z - 1, maxBox.z+1));

				float3 xyz = make_float3(	pos.x + ((Xnew.x - tex1Dfetch(xgrid, pos.x + CUBE_INC)) / (tex1Dfetch(xgrid, pos.x+1 + CUBE_INC) - tex1Dfetch(xgrid, pos.x + CUBE_INC))),
											pos.y + ((Xnew.y - tex1Dfetch(ygrid, pos.y + CUBE_INC)) / (tex1Dfetch(ygrid, pos.y+1 + CUBE_INC) - tex1Dfetch(ygrid, pos.y + CUBE_INC))),
											pos.z + ((Xnew.z - tex1Dfetch(zgrid, pos.z + CUBE_INC)) / (tex1Dfetch(zgrid, pos.z+1 + CUBE_INC) - tex1Dfetch(zgrid, pos.z + CUBE_INC))));

				float3 n = getNormal(xyz, data, minBoxD,  dimD);
				float3 l = Xnew - origin;// ligth; light on the camera
				l = normalize(l);	
				float dif = fabsf(n.x*l.x + n.y*l.y + n.z*l.z);
				float a = Xnew.y/maxHeight;
				int pa = floorf(a*NUM_COLORS);
				if (pa < 0)
				{
					screen[tid*3]   =r[0]*dif;
					screen[tid*3+1] =g[0]*dif;
					screen[tid*3+2] =b[0]*dif;
				}
				else if (pa >= NUM_COLORS-1) 
				{
					screen[tid*3]   = r[NUM_COLORS-1]*dif;
					screen[tid*3+1] = g[NUM_COLORS-1]*dif;
					screen[tid*3+2] = b[NUM_COLORS-1]*dif;
				}
				else
				{
					float dx = (a*(float)NUM_COLORS - (float)pa);
					screen[tid*3]   = (r[pa] + (r[pa+1]-r[pa])*dx)*dif;
					screen[tid*3+1] = (g[pa] + (g[pa+1]-g[pa])*dx)*dif;
					screen[tid*3+2] = (b[pa] + (b[pa+1]-b[pa])*dx)*dif;
				}
				cube->state= CUDA_PAINTED;
				return true;
			}
			else
			{
				cube->state = CUDA_NOCUBE;
				return false;
			}
		}
		else
		{
			screen[tid*3] = 1.0f;
			screen[tid*3+1] = 0.0f;
			screen[tid*3+2] = 0.0f;
			cube->state = CUDA_PAINTED;
			return false;
		}
	}
	return false;
}


__global__ void cuda_getFirtsVoxel(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int levelCube, visibleCube_t * p_indexNode, int numElements, int offset, int3 realDim, float * r, float * g, float * b, float * pixelBuffer, float iso, float maxHeight, float ** tableCubes)
{
	int i = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x +threadIdx.x;

	if (i < numElements)
	{
		float			currentTnear	= 0.0f;
		float			currentTfar		= 0.0f;

		i += offset;
    	int is = i % pvpW;
		int js = i / pvpW;

		float3 ray = LB - origin;
		ray += (js*h)*up + (is*w)*right;
		ray = normalize(ray);

		visibleCube_t * indexNode	= &p_indexNode[i];
		index_node_t minV = coordinateToIndex(make_int3(0,0,0), levelCube, nLevels); 

		if (indexNode->state == CUDA_CACHED)
		{
			_cuda_rayCaster(origin, LB, up, right, w, h, pvpW, pvpH, numElements, iso, indexNode, finalLevel, levelCube, nLevels, maxHeight, realDim, r, g, b, pixelBuffer, offset, tableCubes[indexNode->idCube - minV]);
		}

		if (indexNode->state ==  CUDA_NOCUBE)
		{
			while(1)
			{
				if (_cuda_octreeIteration(octree, sizes, origin, ray, nLevels, finalLevel, indexNode, realDim, &currentTnear, &currentTfar))
				{
					index_node_t idCubeN = indexNode->id >> (3*(finalLevel - levelCube));
					float * d = tableCubes[idCubeN - minV];

					if ( d != 0)
					{
						if (_cuda_rayCaster(origin, LB, up, right, w, h, pvpW, pvpH, numElements, iso, indexNode, finalLevel, levelCube, nLevels, maxHeight, realDim, r, g, b, pixelBuffer, offset, d))
						{
							return;
						}
						else
						{
							indexNode->state = CUDA_NOCUBE;
						}
					}
					else
					{
						indexNode->state = CUDA_CUBE;
						return;
					}
				}
				else
				{
					// NO CUBE FOUND
					pixelBuffer[i*3] = r[NUM_COLORS];
					pixelBuffer[i*3+1] = g[NUM_COLORS];
					pixelBuffer[i*3+2] = b[NUM_COLORS];
					indexNode->state = CUDA_PAINTED;
					return;
				}
			}
		}
	}
}

__global__ void cuda_drawCubes(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int numElements, int3 realDim, float maxHeight, float * r, float * g, float * b, float * screen)
{
	int i = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x +threadIdx.x;

	if (i < numElements)
	{
    	int is = i % pvpW;
		int js = i / pvpW;

		float3 ray = LB - origin;
		ray += (js*h)*up + (is*w)*right;
		ray = normalize(ray);

		float			currentTnear	= 0.0f;
		float			currentTfar		= 0.0f;
		visibleCube_t indexNode = {0};

		// Update tnear and tfar
		if (_cuda_octreeIteration(octree, sizes, origin, ray, nLevels, finalLevel, &indexNode, realDim, &currentTnear, &currentTfar))
		{
			int3 minBox = getMinBoxIndex2(indexNode.id, finalLevel, nLevels);
			int dim = 1 << (3*(nLevels - finalLevel));
			float3 minBoxC = _cuda_BoxToCoordinates(minBox , realDim);
			float3 maxBoxC = _cuda_BoxToCoordinates(minBox + make_int3(dim,dim,dim), realDim);
			float3 n = make_float3(0.0f,0.0f,0.0f);
			float3 hit = origin + ray*currentTnear;

			if(fabsf(hit.x - minBoxC.x) < EPS) 
				n.x = -1.0f;
			else if(fabsf(hit.x - maxBoxC.x) < EPS) 
				n.x = 1.0f;
			else if(fabsf(hit.y - minBoxC.y) < EPS) 
				n.y = -1.0f;
			else if(fabsf(hit.y - maxBoxC.y) < EPS) 
				n.y = 1.0f;
			else if(fabsf(hit.z - minBoxC.z) < EPS) 
				n.z = -1.0f;
			else if(fabsf(hit.z - maxBoxC.z) < EPS) 
				n.z = 1.0f;

			float3 l = hit - origin;// ligth; light on the camera
			l = normalize(l);	
			float dif = fabsf(n.x*l.x + n.y*l.y + n.z*l.z);

			float a = hit.y/maxHeight;
			int pa = floorf(a*NUM_COLORS);
			if (pa < 0)
			{
				screen[i*3]   =r[0]*dif;
				screen[i*3+1] =g[0]*dif;
				screen[i*3+2] =b[0]*dif;
			}
			else if (pa >= NUM_COLORS-1) 
			{
				screen[i*3]   = r[NUM_COLORS-1]*dif;
				screen[i*3+1] = g[NUM_COLORS-1]*dif;
				screen[i*3+2] = b[NUM_COLORS-1]*dif;
			}
			else
			{
				float dx = (a*(float)NUM_COLORS - (float)pa);
				screen[i*3]   = (r[pa] + (r[pa+1]-r[pa])*dx)*dif;
				screen[i*3+1] = (g[pa] + (g[pa+1]-g[pa])*dx)*dif;
				screen[i*3+2] = (b[pa] + (b[pa+1]-b[pa])*dx)*dif;
			}
		}
		else
		{
			// NO CUBE FOUND
			screen[i*3] = r[NUM_COLORS];
			screen[i*3+1] = g[NUM_COLORS];
			screen[i*3+2] = b[NUM_COLORS];
		}
	}
}

/*
 ******************************************************************************************************
 ************ METHODS OCTREEMCUDA *********************************************************************
 ******************************************************************************************************
 */

void getBoxIntersectedOctree(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int levelCube, int numElements, visibleCubeGPU_t visibleGPU, int offset, int3 realDim, float * r, float * g, float * b, float * pixelBuffer, float iso, float maxHeight, float ** tableCubes, hipStream_t stream)
{

	dim3 threads = getThreads(numElements);
	dim3 blocks = getBlocks(numElements);

	cuda_getFirtsVoxel<<<blocks,threads, 0, stream>>>(octree, sizes, nLevels, origin, LB, up, right, w, h, pvpW, pvpH, finalLevel, levelCube, visibleGPU, numElements, offset, realDim, r, g, b, pixelBuffer, iso, maxHeight, tableCubes);

	#ifndef NDEBUG
	if (hipSuccess != hipStreamSynchronize(stream))
	{
		std::cerr<<"Error octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
	#endif

}

	void drawCubes(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int numElements, int3 realDim, float maxHeight, float * r, float * g, float * b, float * screen, hipStream_t stream)
{
	dim3 threads = getThreads(numElements);
	dim3 blocks = getBlocks(numElements);

	cuda_drawCubes<<<blocks,threads, 0, stream>>>(octree, sizes, nLevels, origin, LB, up, right, w, h, pvpW, pvpH, finalLevel, numElements, realDim, maxHeight, r, g, b, screen);

	if (hipSuccess != hipStreamSynchronize(stream))
	{
		std::cerr<<"Error octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
}

/*
 ******************************************************************************************************
 ************ METHODS OCTREE CUDA CREATE **************************************************************
 ******************************************************************************************************
 */

__global__ void cuda_insertOctreePointers(index_node_t ** octreeGPU, int * sizes, index_node_t * memoryGPU)
{
	int offset = 0;
	for(int i=0;i<threadIdx.x; i++)
		offset+=sizes[i];

	octreeGPU[threadIdx.x] = &memoryGPU[offset];
}


void insertOctreePointers(index_node_t ** octreeGPU, int * sizes, index_node_t * memoryGPU, int levels)
{
	dim3 blocks(1);
	dim3 threads(levels);

	cuda_insertOctreePointers<<<blocks,threads,0, 0>>>(octreeGPU, sizes, memoryGPU);

	if (hipSuccess != hipStreamSynchronize(0))
	{
		std::cerr<<"Error init octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
}

}
