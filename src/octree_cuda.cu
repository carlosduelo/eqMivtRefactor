#include "hip/hip_runtime.h"
/*
Author: Carlos Duelo Serrano 
Company: Cesvima

Notes:

*/

#include "octree_cuda.h"

#ifndef DEVICE_CODE
#include <../src/textures.cu>
#endif

#include "cuda_help.h"
#include "mortonCodeUtil.h"

#include "cutil_math.h"

#include <iostream>
#include <fstream>

namespace eqMivt
{
#ifndef DEVICE_CODE
inline __device__ float3 _cuda_BoxToCoordinates(int3 pos, int3 realDim)
{
	float3 r;
	r.x = pos.x >= realDim.x ? tex1Dfetch(xgrid, CUBE_INC + realDim.x-1) : tex1Dfetch(xgrid, CUBE_INC + pos.x);
	r.y = pos.y >= realDim.y ? tex1Dfetch(ygrid, CUBE_INC + realDim.y-1) : tex1Dfetch(ygrid, CUBE_INC + pos.y);
	r.z = pos.z >= realDim.z ? tex1Dfetch(zgrid, CUBE_INC + realDim.z-1) : tex1Dfetch(zgrid, CUBE_INC + pos.z);

	return r;
}
#endif
/*
 **********************************************************************************************
 ****** GPU Octree functions ******************************************************************
 **********************************************************************************************
 */

__device__ inline bool _cuda_checkRangeGrid(index_node_t * elements, index_node_t index, int min, int max)
{
		return elements[max] >= index && elements[min] <= index;
}

__device__ int _cuda_binary_search_closer_Grid(index_node_t * elements, index_node_t index, int min, int max)
{
	int middle = 0;
	while(1)
	{
		int diff 	= max-min;
		middle	= min + (diff / 2);
		if (middle % 2 == 1) middle--;

		if (diff <= 1) return middle;
		if (elements[middle+1] >= index && elements[middle] <= index) return middle;
		if (index < elements[middle])
			max = middle-1;
		else //(index > elements[middle+1])
			min = middle + 2;
	}
}

__device__  bool _cuda_searchSecuentialGrid(index_node_t * elements, index_node_t index, int min, int max)
{
	for(int i=min; i<max; i+=2)
		if (elements[i+1] >= index && elements[i] <= index)
			return true;

	return false;
}

__device__ bool _cuda_RayAABB(index_node_t index, float3 origin, float3 dir,  float * tnear, float * tfar, int nLevels, int3 realDim)
{
	int3 minBoxC;
	int3 maxBoxC;
	int level;
	minBoxC = getMinBoxIndex(index, &level, nLevels); 
	if (minBoxC.x >= realDim.x || minBoxC.y >= realDim.y || minBoxC.y >= realDim.y)
		return false;
	int dim = (1<<(nLevels-level));
	maxBoxC.x = dim + minBoxC.x;
	maxBoxC.y = dim + minBoxC.y;
	maxBoxC.z = dim + minBoxC.z;
	float3 minBox = _cuda_BoxToCoordinates(minBoxC, realDim);
	float3 maxBox = _cuda_BoxToCoordinates(maxBoxC, realDim);

	float tmin, tmax, tymin, tymax, tzmin, tzmax;
	float divx = 1.0f / dir.x;
	if (divx >= 0.0f)
	{
		tmin = (minBox.x - origin.x)*divx;
		tmax = (maxBox.x - origin.x)*divx;
	}
	else
	{
		tmin = (maxBox.x - origin.x)*divx;
		tmax = (minBox.x - origin.x)*divx;
	}
	float divy = 1.0f / dir.y;
	if (divy >= 0.0f)
	{
		tymin = (minBox.y - origin.y)*divy;
		tymax = (maxBox.y - origin.y)*divy;
	}
	else
	{
		tymin = (maxBox.y - origin.y)*divy;
		tymax = (minBox.y - origin.y)*divy;
	}

	if ( (tmin > tymax) || (tymin > tmax) )
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	float divz = 1.0f / dir.z;
	if (divz >= 0.0f)
	{
		tzmin = (minBox.z - origin.z)*divz;
		tzmax = (maxBox.z - origin.z)*divz;
	}
	else
	{
		tzmin = (maxBox.z - origin.z)*divz;
		tzmax = (minBox.z - origin.z)*divz;
	}

	if ( (tmin > tzmax) || (tzmin > tmax) )
		return false;

	if (tzmin > tmin)
		tmin = tzmin;
	if (tzmax < tmax)
		tmax = tzmax;

	if (tmin<0.0f)
	 	*tnear=0.0f;
	else
		*tnear=tmin;
	*tfar=tmax;

	return *tnear < *tfar;
}

__device__ bool _cuda_RayAABB2(float3 origin, float3 dir,  float * tnear, float * tfar, int nLevels, int3 minBoxC, int level, int3 realDim)
{
	if (minBoxC.x >= realDim.x || minBoxC.y >= realDim.y || minBoxC.y >= realDim.y)
		return false;

	int3 maxBoxC;
	int dim = (1<<(nLevels-level));
	maxBoxC.x = dim + minBoxC.x;
	maxBoxC.y = dim + minBoxC.y;
	maxBoxC.z = dim + minBoxC.z;
	float3 minBox = _cuda_BoxToCoordinates(minBoxC, realDim);
	float3 maxBox = _cuda_BoxToCoordinates(maxBoxC, realDim);

	float tmin, tmax, tymin, tymax, tzmin, tzmax;
	float divx = 1.0f / dir.x;
	if (divx >= 0.0f)
	{
		tmin = (minBox.x - origin.x)*divx;
		tmax = (maxBox.x - origin.x)*divx;
	}
	else
	{
		tmin = (maxBox.x - origin.x)*divx;
		tmax = (minBox.x - origin.x)*divx;
	}
	float divy = 1.0f / dir.y;
	if (divy >= 0.0f)
	{
		tymin = (minBox.y - origin.y)*divy;
		tymax = (maxBox.y - origin.y)*divy;
	}
	else
	{
		tymin = (maxBox.y - origin.y)*divy;
		tymax = (minBox.y - origin.y)*divy;
	}

	if ( (tmin > tymax) || (tymin > tmax) )
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	float divz = 1.0f / dir.z;
	if (divz >= 0.0f)
	{
		tzmin = (minBox.z - origin.z)*divz;
		tzmax = (maxBox.z - origin.z)*divz;
	}
	else
	{
		tzmin = (maxBox.z - origin.z)*divz;
		tzmax = (minBox.z - origin.z)*divz;
	}

	if ( (tmin > tzmax) || (tzmin > tmax) )
		return false;
	if (tzmin > tmin)
		tmin = tzmin;
	if (tzmax < tmax)
		tmax = tzmax;

	if (fabsf(tmax -tmin) < EPS)
		return false;

	if (tmin<0.0f)
	 	*tnear=0.0f;
	else
		*tnear=tmin;

	*tfar=tmax;

	return *tnear < *tfar;

}

__device__ int3 _cuda_brother(int3 minBox, index_node_t a, int dim)
{
	int3 r;
	r.z = minBox.z + (a & 0x1) * dim; a>>=1;
	r.y = minBox.y + (a & 0x1) * dim; a>>=1; 
	r.x = minBox.x + (a & 0x1) * dim;

	return r;
}

#if 0
__device__ bool _cuda_searchNextChildrenValidAndHit(index_node_t * elements, int size, int3 realDim, float3 origin, float3 ray, index_node_t father, float cTnear, float cTfar, int nLevels, int level, int3 minBox, index_node_t * child, float * childTnear, float * childTfar)
{
	index_node_t childrenID = father << 3;
	int dim = (1<<(nLevels-level));

	float closer = 0x7ff0000000000000;	//infinity
	bool find = false;
	float childTnearT = 0xfff0000000000000; // -infinity
	float childTfarT  = 0xfff0000000000000; // -infinity

	int closer1 = 0;

	if (size != 2)
	{
		closer1 = _cuda_binary_search_closer_Grid(elements, childrenID,   0, size-1);
	}

	index_node_t lastChildren = childrenID + 7;
	index_node_t min = elements[closer1];
	index_node_t max = elements[closer1+1];

	if (min > lastChildren)
		return false;
	if (min < childrenID)
		min = childrenID;
	if (max > lastChildren)
		max = lastChildren;
	
	while(childrenID <= lastChildren)
	{
		while(childrenID <= max)
		{
			if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, _cuda_brother(minBox, childrenID & 0x7,dim), level, realDim) && childTnearT >= cTnear && childTnearT <= closer)
			{
				*child = childrenID;
				*childTnear = childTnearT;
				*childTfar = childTfarT;
				closer = childTnearT;
				find = true;
			}
			childrenID++;
		}
		closer1+=2;
		if (closer1 >= size)
			return find;
		min = elements[closer1];
		max = elements[closer1+1];
		if (max > lastChildren)
			max = lastChildren; 
		if (min < childrenID)
			min = childrenID;
		childrenID = min;
	}

	return find;
#else
__device__ bool _cuda_searchNextChildrenValidAndHit(index_node_t * elements, int size, int3 realDim, float3 origin, float3 ray, index_node_t father, float cTnear, float cTfar, int nLevels, int level, int3 minB, index_node_t * child, float * childTnear, float * childTfar)
{
	index_node_t childrenID = father << 3;
	int dim = (1<<(nLevels-level));
	int3 minBox = minB;

	float closer = 0x7ff0000000000000;	//infinity
	bool find = false;
	float childTnearT = 0xfff0000000000000; // -infinity
	float childTfarT  = 0xfff0000000000000; // -infinity
	if (size==2)
	{
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.x+=dim;
		minBox.y-=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
	}
	else
	{
		unsigned int closer1 = _cuda_binary_search_closer_Grid(elements, childrenID,   0, size-1);
		unsigned int closer8 = _cuda_binary_search_closer_Grid(elements, childrenID+7, closer1, size-1) + 1;

		if (closer8 >= size)
			closer8 = size-1;

		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.x+=dim;
		minBox.y-=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
	}

	return find;
#endif
}

__device__ int3 _cuda_updateCoordinatesGrid(int maxLevel, int cLevel, index_node_t cIndex, int nLevel, index_node_t nIndex, int3 minBox)
{
	if ( 0 == nIndex)
	{
		return make_int3(0,0,0);
	}
	else if (cLevel < nLevel)
	{
		int dim = 1 << (maxLevel-nLevel);
		minBox.z +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.y +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.x +=  (nIndex & 0x1) * dim;
		return minBox;

	}
	else if (cLevel > nLevel)
	{
		return	getMinBoxIndex2(nIndex, nLevel, maxLevel);
	}
	else
	{
		int dim = 1 << (maxLevel-nLevel);
		minBox.z +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.y +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.x +=  (nIndex & 0x1) * dim;
		minBox.z -=  (cIndex & 0x1) * dim; cIndex>>=1;
		minBox.y -=  (cIndex & 0x1) * dim; cIndex>>=1;
		minBox.x -=  (cIndex & 0x1) * dim;
		return minBox;
	}
}

__device__ bool _cuda_octreeIteration(index_node_t ** octree, int * sizes, float3 origin, float3 ray, int nLevels, int finalLevel, visibleCube_t * indexNode, int3 realDim, float * currentTnear, float * currentTfar)
{
	*currentTnear	= 0.0f;
	*currentTfar	= 0.0f;
	index_node_t 	current			= indexNode->id == 0 ? 1 : indexNode->id;
	int				currentLevel	= 0;

	// Update tnear and tfar
	if (!_cuda_RayAABB(current, origin, ray,  currentTnear, currentTfar, nLevels, realDim) || (*currentTfar) < 0.0f)
	{
		// NO CUBE FOUND
		indexNode->state = CUDA_NOCUBE;
		return false;
	}
	if (current != 1)
	{
		current  >>= 3;
		currentLevel = finalLevel - 1;
		*currentTnear = *currentTfar;
	}

	int3		minBox 		= getMinBoxIndex2(current, currentLevel, nLevels);

	while(1)
	{
		if (currentLevel == finalLevel)
		{
			indexNode->id = current;
			indexNode->state = CUDA_CUBE;
			return true;
		}

		// Get fitst child >= currentTnear away
		index_node_t	child;
		float			childTnear;
		float			childTfar;
		if (_cuda_searchNextChildrenValidAndHit(octree[currentLevel+1], sizes[currentLevel+1], realDim, origin, ray, current, *currentTnear, *currentTfar, nLevels, currentLevel+1, minBox, &child, &childTnear, &childTfar))
		{
			minBox = _cuda_updateCoordinatesGrid(nLevels, currentLevel, current, currentLevel + 1, child, minBox);
			current = child;
			currentLevel++;
			*currentTnear = childTnear;
			*currentTfar = childTfar;
		//if (currentTnear == currentTfar)
		//	printf("--> %d %lld %d %f %f\n", i, current, currentLevel, currentTnear, currentTfar);
		}
		else if (current == 1) 
		{
			indexNode->state = CUDA_NOCUBE;
			return false;
		}
		else
		{
			minBox = _cuda_updateCoordinatesGrid(nLevels, currentLevel, current, currentLevel - 1, current >> 3, minBox);
			current >>= 3;
			currentLevel--;
			*currentTnear = *currentTfar;
		}

	}
}

__device__ bool _cuda_rayCaster(float3 origin, float3  LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int numRays, float iso, visibleCube_t * cube, int levelO, int levelC, int nLevel, float maxHeight, int3 realDim, float * r, float * g, float * b, float * screen, int offset, float * data)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < numRays)
	{
		tid += offset;
		float tnear;
		float tfar;
		// To do test intersection real cube position
		int3 minBox = getMinBoxIndex2(cube->id, levelO, nLevel);
		int dim = 1 << (nLevel-levelO);
		//int dim = powf(2,nLevel-levelO);
		int3 maxBox = minBox + make_int3(dim,dim,dim);
		float3 minBoxC = _cuda_BoxToCoordinates(minBox, realDim);
		float3 maxBoxC = _cuda_BoxToCoordinates(maxBox, realDim);

		int i = tid % pvpW;
		int j = tid / pvpW;

		float3 ray = LB - origin;
		ray += (j*h)*up + (i*w)*right;
		ray = normalize(ray);

		if  (_cuda_RayAABB(origin, ray,  &tnear, &tfar, minBoxC, maxBoxC))
		{
			// To ray caster is needed bigger cube, so add cube inc
			int3 minBoxD = getMinBoxIndex2(cube->id >> (3*(levelO - levelC)), levelC, nLevel) - make_int3(CUBE_INC, CUBE_INC, CUBE_INC);
			int3 dimD;
			dimD.x = (1 << (nLevel-levelC)) + 2*CUBE_INC;
			dimD.y = (1 << (nLevel-levelC)) + 2*CUBE_INC;
			dimD.z = (1 << (nLevel-levelC)) + 2*CUBE_INC;

			float3 Xnear = origin + tnear * ray;

			int3 pos = make_int3(	_cuda_searchCoordinateX(Xnear.x, minBox.x - 1, maxBox.x+1),
									_cuda_searchCoordinateY(Xnear.y, minBox.y - 1, maxBox.y+1),
									_cuda_searchCoordinateZ(Xnear.z, minBox.z - 1, maxBox.z+1));

			bool hit = false;
			float3 Xfar = Xnear;
			float3 Xnew = Xnear;
			bool primera 	= true;
			float ant		= 0.0f;
			float sig		= 0.0f;

			while (!hit &&
				(minBox.x-1 <= pos.x && pos.x <= maxBox.x) &&
				(minBox.y-1 <= pos.y && pos.y <= maxBox.y) &&
				(minBox.z-1 <= pos.z && pos.z <= maxBox.z))
			{
				if (pos.x >= 0 && pos.y >= 0 && pos.z >= 0 && pos.x < realDim.x-1 && pos.y < realDim.y-1 && pos.z < realDim.z-1)
				{
					float3 xyz = make_float3(	pos.x + ((Xnear.x - tex1Dfetch(xgrid, pos.x + CUBE_INC)) / (tex1Dfetch(xgrid, pos.x+1 + CUBE_INC) - tex1Dfetch(xgrid, pos.x + CUBE_INC))),
												pos.y + ((Xnear.y - tex1Dfetch(ygrid, pos.y + CUBE_INC)) / (tex1Dfetch(ygrid, pos.y+1 + CUBE_INC) - tex1Dfetch(ygrid, pos.y + CUBE_INC))),
												pos.z + ((Xnear.z - tex1Dfetch(zgrid, pos.z + CUBE_INC)) / (tex1Dfetch(zgrid, pos.z+1 + CUBE_INC) - tex1Dfetch(zgrid, pos.z + CUBE_INC))));
					
					if (primera)
					{
						ant = getElementInterpolateGrid(xyz, data, minBoxD, dimD);
						Xfar = Xnear;
						primera = false;
					}
					else
					{
						sig = getElementInterpolateGrid(xyz, data, minBoxD, dimD);

						if (( ((iso-ant)<0.0f) && ((iso-sig)<0.0f)) || ( ((iso-ant)>0.0f) && ((iso-sig)>0.0)))
						{
							ant = sig;
							Xfar=Xnear;
						}
						else
						{
							float a = (iso-ant)/(sig-ant);
							Xnew = Xfar*(1.0f-a)+ Xnear*a;
							hit = true;
						}
					}
				}

				// Update Xnear
				Xnear += ((fminf(fabs(tex1Dfetch(xgrid, pos.x+1 + CUBE_INC) - tex1Dfetch(xgrid, pos.x + CUBE_INC)), fminf(fabs(tex1Dfetch(ygrid, pos.y+1 + CUBE_INC) - tex1Dfetch(ygrid, pos.y + CUBE_INC)),fabs( tex1Dfetch(zgrid, pos.z+1 + CUBE_INC) - tex1Dfetch(zgrid, pos.z + CUBE_INC))))) / 3.0f) * ray;

				// Get new pos
				while((minBox.x-2 <= pos.x && pos.x <= maxBox.x + 1) &&  !(tex1Dfetch(xgrid, pos.x + CUBE_INC) <= Xnear.x && Xnear.x < tex1Dfetch(xgrid, pos.x+1 + CUBE_INC)))
					pos.x = ray.x < 0 ? pos.x - 1 : pos.x +1;
				while((minBox.y-2 <= pos.y && pos.y <= maxBox.y + 1) &&!(tex1Dfetch(ygrid, pos.y + CUBE_INC) <= Xnear.y && Xnear.y < tex1Dfetch(ygrid, pos.y+1 + CUBE_INC)))
					pos.y = ray.y < 0 ? pos.y - 1 : pos.y +1;
				while((minBox.z-2 <= pos.z && pos.z <= maxBox.z + 1) &&!(tex1Dfetch(zgrid, pos.z + CUBE_INC) <= Xnear.z && Xnear.z < tex1Dfetch(zgrid, pos.z+1 + CUBE_INC)))
					pos.z = ray.z < 0 ? pos.z - 1 : pos.z +1;
			}

			if (hit)
			{
				pos = make_int3(	_cuda_searchCoordinateX(Xnew.x, minBox.x - 1, maxBox.x+1),
									_cuda_searchCoordinateY(Xnew.y, minBox.y - 1, maxBox.y+1),
									_cuda_searchCoordinateZ(Xnew.z, minBox.z - 1, maxBox.z+1));

				float3 xyz = make_float3(	pos.x + ((Xnew.x - tex1Dfetch(xgrid, pos.x + CUBE_INC)) / (tex1Dfetch(xgrid, pos.x+1 + CUBE_INC) - tex1Dfetch(xgrid, pos.x + CUBE_INC))),
											pos.y + ((Xnew.y - tex1Dfetch(ygrid, pos.y + CUBE_INC)) / (tex1Dfetch(ygrid, pos.y+1 + CUBE_INC) - tex1Dfetch(ygrid, pos.y + CUBE_INC))),
											pos.z + ((Xnew.z - tex1Dfetch(zgrid, pos.z + CUBE_INC)) / (tex1Dfetch(zgrid, pos.z+1 + CUBE_INC) - tex1Dfetch(zgrid, pos.z + CUBE_INC))));

				float3 n = getNormal(xyz, data, minBoxD,  dimD);
				float3 l = Xnew - origin;// ligth; light on the camera
				l = normalize(l);	
				float dif = fabsf(n.x*l.x + n.y*l.y + n.z*l.z);
				float a = Xnew.y/maxHeight;
				int pa = floorf(a*NUM_COLORS);
				if (pa < 0)
				{
					screen[tid*3]   =r[0]*dif;
					screen[tid*3+1] =g[0]*dif;
					screen[tid*3+2] =b[0]*dif;
				}
				else if (pa >= NUM_COLORS-1) 
				{
					screen[tid*3]   = r[NUM_COLORS-1]*dif;
					screen[tid*3+1] = g[NUM_COLORS-1]*dif;
					screen[tid*3+2] = b[NUM_COLORS-1]*dif;
				}
				else
				{
					float dx = (a*(float)NUM_COLORS - (float)pa);
					screen[tid*3]   = (r[pa] + (r[pa+1]-r[pa])*dx)*dif;
					screen[tid*3+1] = (g[pa] + (g[pa+1]-g[pa])*dx)*dif;
					screen[tid*3+2] = (b[pa] + (b[pa+1]-b[pa])*dx)*dif;
				}
				cube->state= CUDA_PAINTED;
				return true;
			}
			else
			{
				cube->state = CUDA_NOCUBE;
				return false;
			}
		}
		else
		{
			screen[tid*3] = 1.0f;
			screen[tid*3+1] = 0.0f;
			screen[tid*3+2] = 0.0f;
			cube->state = CUDA_PAINTED;
			return false;
		}
	}
	return false;
}


__global__ void cuda_getFirtsVoxel(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int levelCube, visibleCube_t * p_indexNode, int numElements, int offset, int3 realDim, float * r, float * g, float * b, float * pixelBuffer, float iso, float maxHeight, float ** tableCubes)
{
	int i = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x +threadIdx.x;

	if (i < numElements)
	{
		float			currentTnear	= 0.0f;
		float			currentTfar		= 0.0f;

		i += offset;
    	int is = i % pvpW;
		int js = i / pvpW;

		float3 ray = LB - origin;
		ray += (js*h)*up + (is*w)*right;
		ray = normalize(ray);

		visibleCube_t * indexNode	= &p_indexNode[i];
		index_node_t minV = coordinateToIndex(make_int3(0,0,0), levelCube, nLevels); 

		if (indexNode->state ==  CUDA_PAINTED)
		{
			indexNode->state = CUDA_DONE;
		}
		else if (indexNode->state == CUDA_CACHED)
		{
			_cuda_rayCaster(origin, LB, up, right, w, h, pvpW, pvpH, numElements, iso, indexNode, finalLevel, levelCube, nLevels, maxHeight, realDim, r, g, b, pixelBuffer, offset, tableCubes[indexNode->idCube - minV]);
		}

		if (indexNode->state ==  CUDA_NOCUBE)
		{
			while(1)
			{
				if (_cuda_octreeIteration(octree, sizes, origin, ray, nLevels, finalLevel, indexNode, realDim, &currentTnear, &currentTfar))
				{
					index_node_t idCubeN = indexNode->id >> (3*(finalLevel - levelCube));
					float * d = tableCubes[idCubeN - minV];

					if ( d != 0)
					{
						if (_cuda_rayCaster(origin, LB, up, right, w, h, pvpW, pvpH, numElements, iso, indexNode, finalLevel, levelCube, nLevels, maxHeight, realDim, r, g, b, pixelBuffer, offset, d))
						{
							return;
						}
						else
						{
							indexNode->state = CUDA_NOCUBE;
						}
					}
					else
					{
						indexNode->state = CUDA_CUBE;
						return;
					}
				}
				else
				{
					// NO CUBE FOUND
					pixelBuffer[i*3] = r[NUM_COLORS];
					pixelBuffer[i*3+1] = g[NUM_COLORS];
					pixelBuffer[i*3+2] = b[NUM_COLORS];
					indexNode->state = CUDA_PAINTED;
					return;
				}
			}
		}
	}
}

__global__ void cuda_drawCubes(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int numElements, int3 realDim, float maxHeight, float * r, float * g, float * b, float * screen)
{
	int i = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x +threadIdx.x;

	if (i < numElements)
	{
    	int is = i % pvpW;
		int js = i / pvpW;

		float3 ray = LB - origin;
		ray += (js*h)*up + (is*w)*right;
		ray = normalize(ray);

		float			currentTnear	= 0.0f;
		float			currentTfar		= 0.0f;
		visibleCube_t indexNode = {0};

		// Update tnear and tfar
		if (_cuda_octreeIteration(octree, sizes, origin, ray, nLevels, finalLevel, &indexNode, realDim, &currentTnear, &currentTfar))
		{
			int3 minBox = getMinBoxIndex2(indexNode.id, finalLevel, nLevels);
			int dim = 1 << (3*(nLevels - finalLevel));
			float3 minBoxC = _cuda_BoxToCoordinates(minBox , realDim);
			float3 maxBoxC = _cuda_BoxToCoordinates(minBox + make_int3(dim,dim,dim), realDim);
			float3 n = make_float3(0.0f,0.0f,0.0f);
			float3 hit = origin + ray*currentTnear;
			float aux = 0.0f;

			if (fabsf(maxBoxC.x - origin.x) < fabsf(minBoxC.x - origin.x))
			{
				aux = minBoxC.x;
				minBoxC.x = maxBoxC.x; 
				maxBoxC.x = aux;
			}
			if (fabsf(maxBoxC.y - origin.y) < fabsf(minBoxC.y - origin.y))
			{
				aux = minBoxC.y;
				minBoxC.y = maxBoxC.y; 
				maxBoxC.y = aux;
			}
			if (fabsf(maxBoxC.z - origin.z) < fabsf(minBoxC.z - origin.z))
			{
				aux = minBoxC.z;
				minBoxC.z = maxBoxC.z; 
				maxBoxC.z = aux;
			}

			if(fabsf(hit.x - minBoxC.x) < EPS) 
				n.x = -1.0f;
			else if(fabsf(hit.x - maxBoxC.x) < EPS) 
				n.x = 1.0f;
			else if(fabsf(hit.y - minBoxC.y) < EPS) 
				n.y = -1.0f;
			else if(fabsf(hit.y - maxBoxC.y) < EPS) 
				n.y = 1.0f;
			else if(fabsf(hit.z - minBoxC.z) < EPS) 
				n.z = -1.0f;
			else if(fabsf(hit.z - maxBoxC.z) < EPS) 
				n.z = 1.0f;


			float3 l = hit - origin;// ligth; light on the camera
			l = normalize(l);	
			float dif = fabsf(n.x*l.x + n.y*l.y + n.z*l.z);

			float a = hit.y/maxHeight;
			int pa = floorf(a*NUM_COLORS);
			if (pa < 0)
			{
				screen[i*3]   =r[0]*dif;
				screen[i*3+1] =g[0]*dif;
				screen[i*3+2] =b[0]*dif;
			}
			else if (pa >= NUM_COLORS-1) 
			{
				screen[i*3]   = r[NUM_COLORS-1]*dif;
				screen[i*3+1] = g[NUM_COLORS-1]*dif;
				screen[i*3+2] = b[NUM_COLORS-1]*dif;
			}
			else
			{
				float dx = (a*(float)NUM_COLORS - (float)pa);
				screen[i*3]   = (r[pa] + (r[pa+1]-r[pa])*dx)*dif;
				screen[i*3+1] = (g[pa] + (g[pa+1]-g[pa])*dx)*dif;
				screen[i*3+2] = (b[pa] + (b[pa+1]-b[pa])*dx)*dif;
			}
		}
		else
		{
			// NO CUBE FOUND
			screen[i*3] = r[NUM_COLORS];
			screen[i*3+1] = g[NUM_COLORS];
			screen[i*3+2] = b[NUM_COLORS];
			return;
		}
	}
}

/*
 ******************************************************************************************************
 ************ METHODS OCTREEMCUDA *********************************************************************
 ******************************************************************************************************
 */

void getBoxIntersectedOctree(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int levelCube, int numElements, visibleCubeGPU_t visibleGPU, int offset, int3 realDim, float * r, float * g, float * b, float * pixelBuffer, float iso, float maxHeight, float ** tableCubes, hipStream_t stream)
{

	dim3 threads = getThreads(numElements);
	dim3 blocks = getBlocks(numElements);

	cuda_getFirtsVoxel<<<blocks,threads, 0, stream>>>(octree, sizes, nLevels, origin, LB, up, right, w, h, pvpW, pvpH, finalLevel, levelCube, visibleGPU, numElements, offset, realDim, r, g, b, pixelBuffer, iso, maxHeight, tableCubes);

	#ifndef NDEBUG
	if (hipSuccess != hipStreamSynchronize(stream))
	{
		std::cerr<<"Error octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
	#endif

}

	void drawCubes(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int numElements, int3 realDim, float maxHeight, float * r, float * g, float * b, float * screen, hipStream_t stream)
{
	dim3 threads = getThreads(numElements);
	dim3 blocks = getBlocks(numElements);

	cuda_drawCubes<<<blocks,threads, 0, stream>>>(octree, sizes, nLevels, origin, LB, up, right, w, h, pvpW, pvpH, finalLevel, numElements, realDim, maxHeight, r, g, b, screen);

	if (hipSuccess != hipStreamSynchronize(stream))
	{
		std::cerr<<"Error octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
}

/*
 ******************************************************************************************************
 ************ METHODS OCTREE CUDA CREATE **************************************************************
 ******************************************************************************************************
 */

__global__ void cuda_insertOctreePointers(index_node_t ** octreeGPU, int * sizes, index_node_t * memoryGPU)
{
	int offset = 0;
	for(int i=0;i<threadIdx.x; i++)
		offset+=sizes[i];

	octreeGPU[threadIdx.x] = &memoryGPU[offset];
}


void insertOctreePointers(index_node_t ** octreeGPU, int * sizes, index_node_t * memoryGPU, int levels)
{
	dim3 blocks(1);
	dim3 threads(levels);

	cuda_insertOctreePointers<<<blocks,threads,0, 0>>>(octreeGPU, sizes, memoryGPU);

	if (hipSuccess != hipStreamSynchronize(0))
	{
		std::cerr<<"Error init octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
}

}
