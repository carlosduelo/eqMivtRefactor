#include "hip/hip_runtime.h"
/*
Author: Carlos Duelo Serrano 
Company: Cesvima

Notes:

*/

#include "octree_cuda.h"

#ifndef DEVICE_CODE
#include <../src/textures.cu>
#endif

#include "cuda_help.h"
#include "mortonCodeUtil.h"

#include "cutil_math.h"

#include <iostream>
#include <fstream>

namespace eqMivt
{
#ifndef DEVICE_CODE
inline __device__ float3 _cuda_BoxToCoordinates(int3 pos, int3 realDim)
{
	float3 r;
	r.x = pos.x >= realDim.x ? tex1Dfetch(xgrid, CUBE_INC + realDim.x-1) : tex1Dfetch(xgrid, CUBE_INC + pos.x);
	r.y = pos.y >= realDim.y ? tex1Dfetch(ygrid, CUBE_INC + realDim.y-1) : tex1Dfetch(ygrid, CUBE_INC + pos.y);
	r.z = pos.z >= realDim.z ? tex1Dfetch(zgrid, CUBE_INC + realDim.z-1) : tex1Dfetch(zgrid, CUBE_INC + pos.z);

	return r;
}
#endif
/*
 **********************************************************************************************
 ****** GPU Octree functions ******************************************************************
 **********************************************************************************************
 */

__device__ inline bool _cuda_checkRangeGrid(index_node_t * elements, index_node_t index, int min, int max)
{
		return elements[max] >= index && elements[min] <= index;
}

__device__ int _cuda_binary_search_closer_Grid(index_node_t * elements, index_node_t index, int min, int max)
{
	int middle = 0;
	while(1)
	{
		int diff 	= max-min;
		middle	= min + (diff / 2);
		if (middle % 2 == 1) middle--;

		if (diff <= 1) return middle;
		if (elements[middle+1] >= index && elements[middle] <= index) return middle;
		if (index < elements[middle])
			max = middle-1;
		else //(index > elements[middle+1])
			min = middle + 2;
	}
}

__device__  bool _cuda_searchSecuentialGrid(index_node_t * elements, index_node_t index, int min, int max)
{
	for(int i=min; i<max; i+=2)
		if (elements[i+1] >= index && elements[i] <= index)
			return true;

	return false;
}

__device__ bool _cuda_RayAABB(index_node_t index, float3 origin, float3 dir,  float * tnear, float * tfar, int nLevels, int3 realDim)
{
	int3 minBoxC;
	int3 maxBoxC;
	int level;
	minBoxC = getMinBoxIndex(index, &level, nLevels); 
	if (minBoxC.x >= realDim.x || minBoxC.y >= realDim.y || minBoxC.y >= realDim.y)
		return false;
	int dim = (1<<(nLevels-level));
	maxBoxC.x = dim + minBoxC.x;
	maxBoxC.y = dim + minBoxC.y;
	maxBoxC.z = dim + minBoxC.z;
	float3 minBox = _cuda_BoxToCoordinates(minBoxC, realDim);
	float3 maxBox = _cuda_BoxToCoordinates(maxBoxC, realDim);

	float tmin, tmax, tymin, tymax, tzmin, tzmax;
	float divx = 1.0f / dir.x;
	if (divx >= 0.0f)
	{
		tmin = (minBox.x - origin.x)*divx;
		tmax = (maxBox.x - origin.x)*divx;
	}
	else
	{
		tmin = (maxBox.x - origin.x)*divx;
		tmax = (minBox.x - origin.x)*divx;
	}
	float divy = 1.0f / dir.y;
	if (divy >= 0.0f)
	{
		tymin = (minBox.y - origin.y)*divy;
		tymax = (maxBox.y - origin.y)*divy;
	}
	else
	{
		tymin = (maxBox.y - origin.y)*divy;
		tymax = (minBox.y - origin.y)*divy;
	}

	if ( (tmin > tymax) || (tymin > tmax) )
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	float divz = 1.0f / dir.z;
	if (divz >= 0.0f)
	{
		tzmin = (minBox.z - origin.z)*divz;
		tzmax = (maxBox.z - origin.z)*divz;
	}
	else
	{
		tzmin = (maxBox.z - origin.z)*divz;
		tzmax = (minBox.z - origin.z)*divz;
	}

	if ( (tmin > tzmax) || (tzmin > tmax) )
		return false;

	if (tzmin > tmin)
		tmin = tzmin;
	if (tzmax < tmax)
		tmax = tzmax;

	if (tmin<0.0f)
	 	*tnear=0.0f;
	else
		*tnear=tmin;
	*tfar=tmax;

	return *tnear < *tfar;
}

__device__ bool _cuda_RayAABB2(float3 origin, float3 dir,  float * tnear, float * tfar, int nLevels, int3 minBoxC, int level, int3 realDim)
{
	if (minBoxC.x >= realDim.x || minBoxC.y >= realDim.y || minBoxC.y >= realDim.y)
		return false;

	int3 maxBoxC;
	int dim = (1<<(nLevels-level));
	maxBoxC.x = dim + minBoxC.x;
	maxBoxC.y = dim + minBoxC.y;
	maxBoxC.z = dim + minBoxC.z;
	float3 minBox = _cuda_BoxToCoordinates(minBoxC, realDim);
	float3 maxBox = _cuda_BoxToCoordinates(maxBoxC, realDim);

	float tmin, tmax, tymin, tymax, tzmin, tzmax;
	float divx = 1.0f / dir.x;
	if (divx >= 0.0f)
	{
		tmin = (minBox.x - origin.x)*divx;
		tmax = (maxBox.x - origin.x)*divx;
	}
	else
	{
		tmin = (maxBox.x - origin.x)*divx;
		tmax = (minBox.x - origin.x)*divx;
	}
	float divy = 1.0f / dir.y;
	if (divy >= 0.0f)
	{
		tymin = (minBox.y - origin.y)*divy;
		tymax = (maxBox.y - origin.y)*divy;
	}
	else
	{
		tymin = (maxBox.y - origin.y)*divy;
		tymax = (minBox.y - origin.y)*divy;
	}

	if ( (tmin > tymax) || (tymin > tmax) )
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	float divz = 1.0f / dir.z;
	if (divz >= 0.0f)
	{
		tzmin = (minBox.z - origin.z)*divz;
		tzmax = (maxBox.z - origin.z)*divz;
	}
	else
	{
		tzmin = (maxBox.z - origin.z)*divz;
		tzmax = (minBox.z - origin.z)*divz;
	}

	if ( (tmin > tzmax) || (tzmin > tmax) )
		return false;
	if (tzmin > tmin)
		tmin = tzmin;
	if (tzmax < tmax)
		tmax = tzmax;

	if (fabsf(tmax -tmin) < EPS)
		return false;

	if (tmin<0.0f)
	 	*tnear=0.0f;
	else
		*tnear=tmin;

	*tfar=tmax;

	return *tnear < *tfar;

}

__device__ int3 _cuda_brother(int3 minBox, index_node_t a, int dim)
{
	int3 r;
	r.z = minBox.z + (a & 0x1) * dim; a>>=1;
	r.y = minBox.y + (a & 0x1) * dim; a>>=1; 
	r.x = minBox.x + (a & 0x1) * dim;

	return r;
}

#if 0
__device__ bool _cuda_searchNextChildrenValidAndHit(index_node_t * elements, int size, int3 realDim, float3 origin, float3 ray, index_node_t father, float cTnear, float cTfar, int nLevels, int level, int3 minBox, index_node_t * child, float * childTnear, float * childTfar)
{
	index_node_t childrenID = father << 3;
	int dim = (1<<(nLevels-level));

	float closer = 0x7ff0000000000000;	//infinity
	bool find = false;
	float childTnearT = 0xfff0000000000000; // -infinity
	float childTfarT  = 0xfff0000000000000; // -infinity

	int closer1 = 0;

	if (size != 2)
	{
		closer1 = _cuda_binary_search_closer_Grid(elements, childrenID,   0, size-1);
	}

	index_node_t lastChildren = childrenID + 7;
	index_node_t min = elements[closer1];
	index_node_t max = elements[closer1+1];

	if (min > lastChildren)
		return false;
	if (min < childrenID)
		min = childrenID;
	if (max > lastChildren)
		max = lastChildren;
	
	while(childrenID <= lastChildren)
	{
		while(childrenID <= max)
		{
			if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, _cuda_brother(minBox, childrenID & 0x7,dim), level, realDim) && childTnearT >= cTnear && childTnearT <= closer)
			{
				*child = childrenID;
				*childTnear = childTnearT;
				*childTfar = childTfarT;
				closer = childTnearT;
				find = true;
			}
			childrenID++;
		}
		closer1+=2;
		if (closer1 >= size)
			return find;
		min = elements[closer1];
		max = elements[closer1+1];
		if (max > lastChildren)
			max = lastChildren; 
		if (min < childrenID)
			min = childrenID;
		childrenID = min;
	}

	return find;
#else
__device__ bool _cuda_searchNextChildrenValidAndHit(index_node_t * elements, int size, int3 realDim, float3 origin, float3 ray, index_node_t father, float cTnear, float cTfar, int nLevels, int level, int3 minB, index_node_t * child, float * childTnear, float * childTfar)
{
	index_node_t childrenID = father << 3;
	int dim = (1<<(nLevels-level));
	int3 minBox = minB;

	float closer = 0x7ff0000000000000;	//infinity
	bool find = false;
	float childTnearT = 0xfff0000000000000; // -infinity
	float childTfarT  = 0xfff0000000000000; // -infinity
	if (size==2)
	{
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.x+=dim;
		minBox.y-=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) && childTnearT >= cTnear && childTnearT <= closer &&
			_cuda_checkRangeGrid(elements, childrenID,0,1))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
	}
	else
	{
		unsigned int closer1 = _cuda_binary_search_closer_Grid(elements, childrenID,   0, size-1);
		unsigned int closer8 = _cuda_binary_search_closer_Grid(elements, childrenID+7, closer1, size-1) + 1;

		if (closer8 >= size)
			closer8 = size-1;

		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.x+=dim;
		minBox.y-=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.y+=dim;
		minBox.z-=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
		minBox.z+=dim;
		if (_cuda_RayAABB2(origin, ray,  &childTnearT, &childTfarT, nLevels, minBox, level, realDim) &&  childTnearT>=cTnear && childTnearT <= closer &&
			_cuda_searchSecuentialGrid(elements, childrenID, closer1, closer8))
		{
			*child = childrenID;
			*childTnear = childTnearT;
			*childTfar = childTfarT;
			closer = childTnearT;
			find = true;
		}
		childrenID++;
	}

	return find;
#endif
}

__device__ int3 _cuda_updateCoordinatesGrid(int maxLevel, int cLevel, index_node_t cIndex, int nLevel, index_node_t nIndex, int3 minBox)
{
	if ( 0 == nIndex)
	{
		return make_int3(0,0,0);
	}
	else if (cLevel < nLevel)
	{
		int dim = 1 << (maxLevel-nLevel);
		minBox.z +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.y +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.x +=  (nIndex & 0x1) * dim;
		return minBox;

	}
	else if (cLevel > nLevel)
	{
		return	getMinBoxIndex2(nIndex, nLevel, maxLevel);
	}
	else
	{
		int dim = 1 << (maxLevel-nLevel);
		minBox.z +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.y +=  (nIndex & 0x1) * dim; nIndex>>=1;
		minBox.x +=  (nIndex & 0x1) * dim;
		minBox.z -=  (cIndex & 0x1) * dim; cIndex>>=1;
		minBox.y -=  (cIndex & 0x1) * dim; cIndex>>=1;
		minBox.x -=  (cIndex & 0x1) * dim;
		return minBox;
	}
}

__global__ void cuda_getFirtsVoxel(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, visibleCube_t * p_indexNode, int numElements, int offset, int3 realDim)
{
	int i = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x +threadIdx.x;

	if (i < numElements)
	{
		i += offset;
    	int is = i % pvpW;
		int js = i / pvpW;

		float3 ray = LB - origin;
		ray += (js*h)*up + (is*w)*right;
		ray = normalize(ray);

		visibleCube_t * indexNode	= &p_indexNode[i];

		if (indexNode->state ==  CUDA_PAINTED)
		{
			indexNode->state = CUDA_DONE;
			return;
		}
		else if (indexNode->state ==  CUDA_NOCUBE)
		{
			float			currentTnear	= 0.0f;
			float			currentTfar		= 0.0f;
			index_node_t 	current			= indexNode->id == 0 ? 1 : indexNode->id;
			int				currentLevel	= 0;

			// Update tnear and tfar
			if (!_cuda_RayAABB(current, origin, ray,  &currentTnear, &currentTfar, nLevels, realDim) || currentTfar < 0.0f)
			{
				// NO CUBE FOUND
				indexNode->id 	= 0;
				return;
			}
			if (current != 1)
			{
				current  >>= 3;
				currentLevel = finalLevel - 1;
				currentTnear = currentTfar;
				//printf("--> %d %lld %d %f %f\n", i, current, currentLevel, currentTnear, currentTfar);
			}

			int3		minBox 		= getMinBoxIndex2(current, currentLevel, nLevels);

			while(1)
			{
				if (currentLevel == finalLevel)
				{
					indexNode->id = current;
					indexNode->state = CUDA_CUBE;
					return;
				}

				// Get fitst child >= currentTnear away
				index_node_t	child;
				float			childTnear;
				float			childTfar;
				if (_cuda_searchNextChildrenValidAndHit(octree[currentLevel+1], sizes[currentLevel+1], realDim, origin, ray, current, currentTnear, currentTfar, nLevels, currentLevel+1, minBox, &child, &childTnear, &childTfar))
				{
					minBox = _cuda_updateCoordinatesGrid(nLevels, currentLevel, current, currentLevel + 1, child, minBox);
					current = child;
					currentLevel++;
					currentTnear = childTnear;
					currentTfar = childTfar;
				//if (currentTnear == currentTfar)
				//	printf("--> %d %lld %d %f %f\n", i, current, currentLevel, currentTnear, currentTfar);
				}
				else if (current == 1) 
				{
					indexNode->id 	= 0;
					return;
				}
				else
				{
					minBox = _cuda_updateCoordinatesGrid(nLevels, currentLevel, current, currentLevel - 1, current >> 3, minBox);
					current >>= 3;
					currentLevel--;
					currentTnear = currentTfar;
				}

			}
		}
	}
	return;
}

__global__ void cuda_drawCubes(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int numElements, int3 realDim, float maxHeight, float * r, float * g, float * b, float * screen)
{
	int i = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x +threadIdx.x;

	if (i < numElements)
	{
    	int is = i % pvpW;
		int js = i / pvpW;

		float3 ray = LB - origin;
		ray += (js*h)*up + (is*w)*right;
		ray = normalize(ray);

		float			currentTnear	= 0.0f;
		float			currentTfar		= 0.0f;
		index_node_t 	current			= 1;
		int				currentLevel	= 0;

		// Update tnear and tfar
		if (!_cuda_RayAABB(current, origin, ray,  &currentTnear, &currentTfar, nLevels, realDim) || currentTfar < 0.0f)
		{
			// NO CUBE FOUND
			screen[i*3] = r[NUM_COLORS];
			screen[i*3+1] = g[NUM_COLORS];
			screen[i*3+2] = b[NUM_COLORS];
			return;
		}
		if (current != 1)
		{
			current  >>= 3;
			currentLevel = finalLevel - 1;
			currentTnear = currentTfar;
		}

		int3		minBox 		= getMinBoxIndex2(current, currentLevel, nLevels);

		while(currentLevel != finalLevel)
		{
			// Get fitst child >= currentTnear away
			index_node_t	child;
			float			childTnear;
			float			childTfar;
			if (_cuda_searchNextChildrenValidAndHit(octree[currentLevel+1], sizes[currentLevel+1], realDim, origin, ray, current, currentTnear, currentTfar, nLevels, currentLevel+1, minBox, &child, &childTnear, &childTfar))
			{
				minBox = _cuda_updateCoordinatesGrid(nLevels, currentLevel, current, currentLevel + 1, child, minBox);
				current = child;
				currentLevel++;
				currentTnear = childTnear;
				currentTfar = childTfar;
			}
			else if (current == 1) 
			{
				screen[i*3] = r[NUM_COLORS];
				screen[i*3+1] = g[NUM_COLORS];
				screen[i*3+2] = b[NUM_COLORS];
				return;
			}
			else
			{
				minBox = _cuda_updateCoordinatesGrid(nLevels, currentLevel, current, currentLevel - 1, current >> 3, minBox);
				current >>= 3;
				currentLevel--;
				currentTnear = currentTfar;
			}

		}
		int dim = 1 << (3*(nLevels - currentLevel));
		float3 minBoxC = _cuda_BoxToCoordinates(minBox , realDim);
		float3 maxBoxC = _cuda_BoxToCoordinates(minBox + make_int3(dim,dim,dim), realDim);
		float3 n = make_float3(0.0f,0.0f,0.0f);
		float3 hit = origin + ray*currentTnear;
		float aux = 0.0f;

		if (fabsf(maxBoxC.x - origin.x) < fabsf(minBoxC.x - origin.x))
		{
			aux = minBoxC.x;
			minBoxC.x = maxBoxC.x; 
			maxBoxC.x = aux;
		}
		if (fabsf(maxBoxC.y - origin.y) < fabsf(minBoxC.y - origin.y))
		{
			aux = minBoxC.y;
			minBoxC.y = maxBoxC.y; 
			maxBoxC.y = aux;
		}
		if (fabsf(maxBoxC.z - origin.z) < fabsf(minBoxC.z - origin.z))
		{
			aux = minBoxC.z;
			minBoxC.z = maxBoxC.z; 
			maxBoxC.z = aux;
		}

		if(fabsf(hit.x - minBoxC.x) < EPS) 
			n.x = -1.0f;
		else if(fabsf(hit.x - maxBoxC.x) < EPS) 
			n.x = 1.0f;
		else if(fabsf(hit.y - minBoxC.y) < EPS) 
			n.y = -1.0f;
		else if(fabsf(hit.y - maxBoxC.y) < EPS) 
			n.y = 1.0f;
		else if(fabsf(hit.z - minBoxC.z) < EPS) 
			n.z = -1.0f;
		else if(fabsf(hit.z - maxBoxC.z) < EPS) 
			n.z = 1.0f;


		float3 l = hit - origin;// ligth; light on the camera
		l = normalize(l);	
		float dif = fabsf(n.x*l.x + n.y*l.y + n.z*l.z);

		float a = hit.y/maxHeight;
		int pa = floorf(a*NUM_COLORS);
		if (pa < 0)
		{
			screen[i*3]   =r[0]*dif;
			screen[i*3+1] =g[0]*dif;
			screen[i*3+2] =b[0]*dif;
		}
		else if (pa >= NUM_COLORS-1) 
		{
			screen[i*3]   = r[NUM_COLORS-1]*dif;
			screen[i*3+1] = g[NUM_COLORS-1]*dif;
			screen[i*3+2] = b[NUM_COLORS-1]*dif;
		}
		else
		{
			float dx = (a*(float)NUM_COLORS - (float)pa);
			screen[i*3]   = (r[pa] + (r[pa+1]-r[pa])*dx)*dif;
			screen[i*3+1] = (g[pa] + (g[pa+1]-g[pa])*dx)*dif;
			screen[i*3+2] = (b[pa] + (b[pa+1]-b[pa])*dx)*dif;
		}
	}
}

/*
 ******************************************************************************************************
 ************ METHODS OCTREEMCUDA *********************************************************************
 ******************************************************************************************************
 */

	void getBoxIntersectedOctree(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int numElements, visibleCubeGPU_t visibleGPU, int offset, int3 realDim, hipStream_t stream)
{

	dim3 threads = getThreads(numElements);
	dim3 blocks = getBlocks(numElements);

	cuda_getFirtsVoxel<<<blocks,threads, 0, stream>>>(octree, sizes, nLevels, origin, LB, up, right, w, h, pvpW, pvpH, finalLevel, visibleGPU, numElements, offset, realDim);

	#ifndef NDEBUG
	if (hipSuccess != hipStreamSynchronize(stream))
	{
		std::cerr<<"Error octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
	#endif

}

	void drawCubes(index_node_t ** octree, int * sizes, int nLevels, float3 origin, float3 LB, float3 up, float3 right, float w, float h, int pvpW, int pvpH, int finalLevel, int numElements, int3 realDim, float maxHeight, float * r, float * g, float * b, float * screen, hipStream_t stream)
{
	dim3 threads = getThreads(numElements);
	dim3 blocks = getBlocks(numElements);

	cuda_drawCubes<<<blocks,threads, 0, stream>>>(octree, sizes, nLevels, origin, LB, up, right, w, h, pvpW, pvpH, finalLevel, numElements, realDim, maxHeight, r, g, b, screen);

	if (hipSuccess != hipStreamSynchronize(stream))
	{
		std::cerr<<"Error octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
}

/*
 ******************************************************************************************************
 ************ METHODS OCTREE CUDA CREATE **************************************************************
 ******************************************************************************************************
 */

__global__ void cuda_insertOctreePointers(index_node_t ** octreeGPU, int * sizes, index_node_t * memoryGPU)
{
	int offset = 0;
	for(int i=0;i<threadIdx.x; i++)
		offset+=sizes[i];

	octreeGPU[threadIdx.x] = &memoryGPU[offset];
}


void insertOctreePointers(index_node_t ** octreeGPU, int * sizes, index_node_t * memoryGPU, int levels)
{
	dim3 blocks(1);
	dim3 threads(levels);

	cuda_insertOctreePointers<<<blocks,threads,0, 0>>>(octreeGPU, sizes, memoryGPU);

	if (hipSuccess != hipStreamSynchronize(0))
	{
		std::cerr<<"Error init octree: "<<hipGetErrorString(hipGetLastError())<<std::endl;
		throw;
	}
}

}
