#include "hip/hip_runtime.h"
/*
Author: Carlos Duelo Serrano 
Company: Cesvima

Notes:

*/

#define BLOCK_SIZE 128

#include "createOctree_cuda.h"
#include "mortonCodeUtil.h"
#include "cuda_help.h"
#include <cutil_math.h>

#include "hip/hip_runtime.h"

#include <iostream>

#define posToIndex(i,j,k,d) ((k)+(j)*(d)+(i)*(d)*(d))

namespace eqMivt
{

__device__ bool cuda_checkIsosurface(int x, int y, int z, int dim, float * cube, float isosurface)
{
	bool sign = (cube[posToIndex(x, y, z, dim)] - isosurface) < 0;

	if (((cube[posToIndex(x, y, z+1, dim)] - isosurface) < 0) != sign)
		return true;
	if (((cube[posToIndex(x, y+1, z, dim)] - isosurface) < 0) != sign)
		return true;
	if (((cube[posToIndex(x, y+1, z+1, dim)] - isosurface) < 0) != sign)
		return true;
	if (((cube[posToIndex(x+1, y, z, dim)] - isosurface) < 0) != sign)
		return true;
	if (((cube[posToIndex(x+1, y, z+1, dim)] - isosurface) < 0) != sign)
		return true;
	if (((cube[posToIndex(x+1, y+1, z, dim)] - isosurface) < 0) != sign)
		return true;
	if (((cube[posToIndex(x+1, y+1, z+1, dim)] - isosurface) < 0) != sign)
		return true;

	return false;
}

__global__ void cuda_extracIsosurface(unsigned int numElements, unsigned int cubeLevel, unsigned int nLevels, float iso, index_node_t idCube, unsigned char * result, float * cube)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < numElements)
	{

		int3 coorCubeStart = getMinBoxIndex2(idCube, cubeLevel, nLevels);

		index_node_t id = (idCube << (3*(nLevels - cubeLevel))) + tid;
		int3 coord = getMinBoxIndex2(id, nLevels, nLevels) + make_int3(CUBE_INC, CUBE_INC, CUBE_INC);
		coord -= coorCubeStart;

		int cubeDim = (1 << (nLevels - cubeLevel))  + 2 * CUBE_INC;

		if ( cuda_checkIsosurface(coord.x, coord.y, coord.z, cubeDim, cube, iso))
		{
			result[tid] = (unsigned char)1;
		}
		else
		{	
			result[tid] = (unsigned char)0;
		}
	}
}

void extracIsosurface(unsigned int numElements, unsigned int cubeLevel, unsigned int nLevels, float iso, index_node_t idCube, unsigned char * result, float * cube, hipStream_t stream)
{
	dim3 threads = getThreads(numElements);
	dim3 blocks = getBlocks(numElements);
	cuda_extracIsosurface<<<blocks, threads, 0 , stream>>>(numElements, cubeLevel, nLevels, iso, idCube, result, cube);
}

}
