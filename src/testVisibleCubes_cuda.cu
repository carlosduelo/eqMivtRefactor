#include "hip/hip_runtime.h"
/*
Author: Carlos Duelo Serrano 
Company: Cesvima

Notes:

*/

#include <testVisibleCubes_CUDA.h>
#include <cuda_help.h>

#include <hip/hip_runtime.h>

__global__ void cuda_updateCubesGPU(eqMivt::visibleCubeGPU cubes, int size, eqMivt::statusCube status)
{
	int idx = blockIdx.y * blockDim.x * gridDim.y + blockIdx.x * blockDim.x +threadIdx.x;
	
	if (idx < size)
	{
		cubes[idx].state = status;
	}

	return;
}

void test_updateCubesGPU(eqMivt::visibleCubeGPU cubes, int size, eqMivt::statusCube status)
{
	dim3 threads = eqMivt::getThreads(size);
	dim3 blocks = eqMivt::getBlocks(size);

	cuda_updateCubesGPU<<<blocks,threads, 0, 0>>>(cubes, size, status);
}
