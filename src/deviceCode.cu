#include "hip/hip_runtime.h"
/*
Author: Carlos Duelo Serrano 
Company: Cesvima

Notes:

*/

#define DEVICE_CODE

#include <typedef.h>
#include <../src/textures.cu>

namespace eqMivt
{

inline __device__ float3 _cuda_BoxToCoordinates(int3 pos, int3 realDim)
{
	float3 r;
	r.x = pos.x >= realDim.x ? tex1Dfetch(xgrid, CUBE_INC + realDim.x-1) : tex1Dfetch(xgrid, CUBE_INC + pos.x);
	r.y = pos.y >= realDim.y ? tex1Dfetch(ygrid, CUBE_INC + realDim.y-1) : tex1Dfetch(ygrid, CUBE_INC + pos.y);
	r.z = pos.z >= realDim.z ? tex1Dfetch(zgrid, CUBE_INC + realDim.z-1) : tex1Dfetch(zgrid, CUBE_INC + pos.z);

	return r;
}

}

#include <../src/rayCaster_cuda.cu>
#include <../src/octree_cuda.cu>


