#include "hip/hip_runtime.h"
/*
Author: Carlos Duelo Serrano 
Company: Cesvima

Notes:

*/

#include <textures.h>

#include <hip/hip_runtime.h>

#include <iostream>

#ifndef EQ_MIVT_TEXTURE_H
#define EQ_MIVT_TEXTURE_H

namespace eqMivt
{
	hipChannelFormatDesc XchannelDesc;
	hipChannelFormatDesc YchannelDesc;
	hipChannelFormatDesc ZchannelDesc;

	texture<float> xgrid;
	texture<float> ygrid;
	texture<float> zgrid;

	bool initTextures()
	{
		XchannelDesc = hipCreateChannelDesc<float>();
		YchannelDesc = hipCreateChannelDesc<float>();
		ZchannelDesc = hipCreateChannelDesc<float>();
		return true;
	}

	bool bindTextures(float * xGrid, float * yGrid, float * zGrid, int3 realDim)
	{
		if (hipSuccess != hipBindTexture(NULL, &xgrid, (void*)xGrid, &XchannelDesc, (2*CUBE_INC + realDim.x)*sizeof(float)) ||
			hipSuccess != hipBindTexture(NULL, &ygrid, (void*)yGrid, &YchannelDesc, (2*CUBE_INC + realDim.y)*sizeof(float)) ||
			hipSuccess != hipBindTexture(NULL, &zgrid, (void*)zGrid, &ZchannelDesc, (2*CUBE_INC + realDim.z)*sizeof(float)))
			{
				std::cerr<<"Error binding texture memory: "<<hipGetErrorString(hipGetLastError())<<std::endl;
				return false;
			}

		return true;
	}

	bool unBindTextures()
	{
		if (hipSuccess != hipUnbindTexture(xgrid) || 
			hipSuccess != hipUnbindTexture(ygrid) || 
			hipSuccess != hipUnbindTexture(zgrid))
		{
			std::cerr<<"Error unbinding texture memory: "<<hipGetErrorString(hipGetLastError())<<std::endl;
			return false;
		}
		return true;
	}
}
#endif /* EQ_MIVT_TEXTURE_H*/
