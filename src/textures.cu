#include "hip/hip_runtime.h"
/*
Author: Carlos Duelo Serrano 
Company: Cesvima

Notes:

*/

#include <textures.h>

#include <hip/hip_runtime.h>

#include <iostream>

#ifndef EQ_MIVT_TEXTURE_H
#define EQ_MIVT_TEXTURE_H

namespace eqMivt
{
	hipChannelFormatDesc XchannelDesc;
	hipChannelFormatDesc YchannelDesc;
	hipChannelFormatDesc ZchannelDesc;
	size_t offsetX;
	size_t offsetY;
	size_t offsetZ;

	texture<float, hipTextureType1D, hipReadModeElementType> xgrid;
	texture<float, hipTextureType1D, hipReadModeElementType> ygrid;
	texture<float, hipTextureType1D, hipReadModeElementType> zgrid;

	bool initTextures()
	{
		XchannelDesc = hipCreateChannelDesc<float>();
		YchannelDesc = hipCreateChannelDesc<float>();
		ZchannelDesc = hipCreateChannelDesc<float>();
		offsetX = 0;
		offsetY = 0;
		offsetZ = 0;
		return true;
	}

	bool bindTextures(float * xGrid, float * yGrid, float * zGrid, int3 realDim)
	{
		offsetX = 0;
		offsetY = 0;
		offsetZ = 0;

		if (hipSuccess != hipBindTexture(&offsetX, &xgrid, (void*)xGrid, &XchannelDesc, (2*CUBE_INC + realDim.x)*sizeof(float)) ||
			hipSuccess != hipBindTexture(&offsetY, &ygrid, (void*)yGrid, &YchannelDesc, (2*CUBE_INC + realDim.y)*sizeof(float)) ||
			hipSuccess != hipBindTexture(&offsetZ, &zgrid, (void*)zGrid, &ZchannelDesc, (2*CUBE_INC + realDim.z)*sizeof(float)))
			{
				std::cerr<<"Error binding texture memory: "<<hipGetErrorString(hipGetLastError())<<std::endl;
				return false;
			}

		return true;
	}

	bool unBindTextures()
	{
		if (hipSuccess != hipUnbindTexture(xgrid) || 
			hipSuccess != hipUnbindTexture(ygrid) || 
			hipSuccess != hipUnbindTexture(zgrid))
		{
			std::cerr<<"Error unbinding texture memory: "<<hipGetErrorString(hipGetLastError())<<std::endl;
			return false;
		}
		return true;
	}
}
#endif /* EQ_MIVT_TEXTURE_H*/
